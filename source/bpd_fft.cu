#include "hip/hip_runtime.h"
#include "../headers/bpd_cudacommon.cuh"
#include "../headers/bpd_fft.cuh"

unsigned realToComplexFFT(float* idata, long inputElements, hipfftComplex **odata, int batch)
{
	// ---------------------- Initialise CUDA stuff ---------------------------------
	hipSetDevice(0);
	hipEvent_t event_start, event_stop;
	float timestamp;

	//  Events
	hipEventCreate(&event_start);
	hipEventCreate(&event_stop);

	// FFT size.
	unsigned inputSize = inputElements * sizeof(hipfftReal);
	unsigned outputElements = (floor(inputElements/2) + 1);
	unsigned outputSize = outputElements * sizeof(hipfftComplex);

	// Init variables.
	hipfftHandle plan;
	hipfftReal *device_idata, *host_idata;
	hipfftComplex *device_odata, *host_odata;
	host_idata = (hipfftReal*) malloc(inputSize);
	host_odata = (hipfftComplex *) malloc(outputSize);

	// ---------------------- Copy data to GPU ----------------------------------
	hipMalloc((void **) &device_idata, inputSize);
	hipMalloc((void **) &device_odata, outputSize);
	checkCudaErrors("Allocating memory on device");


	host_idata = (hipfftReal*) idata;

	hipEventRecord(event_start, 0);
	hipMemcpy(device_idata, host_idata, inputSize, hipMemcpyHostToDevice);
	checkCudaErrors("Copying data to device");
	hipEventRecord(event_stop, 0);
	hipEventSynchronize(event_stop);
	hipEventElapsedTime(&timestamp, event_start, event_stop);
	checkCudaErrors("Copying data to device timer");
	//printf("Copied to GPU in: %lf\n", timestamp);

	// ---------------------- FFT all the channels in place ----------------------
	hipfftPlan1d(&plan, inputElements, HIPFFT_R2C, batch);

	hipEventRecord(event_start, 0);
	hipfftExecR2C(plan, device_idata, device_odata);
	hipDeviceSynchronize();
	hipEventRecord(event_stop, 0);
	hipEventSynchronize(event_stop);
	hipEventElapsedTime(&timestamp, event_start, event_stop);
	checkCudaErrors("Performing FFT");
	//printf("Performed forward FFT: %lf\n", timestamp);

	hipMemcpy(host_odata, device_odata, outputSize, hipMemcpyDeviceToHost);

	*odata = host_odata;

//_________________________________________________________________________________________________________
// Perform IFFT.
//hipfftHandle testPlan;
//hipfftPlan1d(&testPlan, inputSamples, HIPFFT_C2R, 1);
//hipfftExecC2R(testPlan, device_odata, device_idata);
//checkCudaErrors("Performing IFFT");

//hipMemcpy(host_idata, device_idata, outputSize, hipMemcpyDeviceToHost);

//_________________________________________________________________________________________________________


	// ---------------------- Cuda Memory Cleanup ----------------------
	hipfftDestroy(plan);
	hipFree(device_idata);
	hipFree(device_odata);

	return outputElements;
}

unsigned complexToComplexFFT(hipfftComplex* device_idata, unsigned int inputElements, hipfftComplex **device_odata, int batch)
{
	// ---------------------- Initialise CUDA stuff ---------------------------------
	hipSetDevice(0);
	hipEvent_t event_start, event_stop;
	float timestamp;

	//  Events
	hipEventCreate(&event_start);
	hipEventCreate(&event_stop);

	// FFT size.
	unsigned int outputElements = inputElements;
	unsigned int outputSize = outputElements * sizeof(hipfftComplex) * batch;

	// Init variables.
	hipfftHandle plan;
	hipfftComplex *odata;
	hipMalloc((void **) &odata, outputSize);
	checkCudaErrors("Allocating memory on device");


	// ---------------------- FFT all the channels in place ----------------------
	hipfftPlan1d(&plan, inputElements, HIPFFT_C2C, batch);

	hipEventRecord(event_start, 0);
	hipfftExecC2C(plan, device_idata, odata,HIPFFT_FORWARD);
	hipDeviceSynchronize();
	hipEventRecord(event_stop, 0);
	hipEventSynchronize(event_stop);
	hipEventElapsedTime(&timestamp, event_start, event_stop);
	checkCudaErrors("Performing FFT");
	//printf("Performed forward FFT: %lf\n", timestamp);

	*device_odata = odata;

//_________________________________________________________________________________________________________
// Perform IFFT.
//hipfftHandle testPlan;
//hipfftPlan1d(&testPlan, inputSamples, HIPFFT_C2R, 1);
//hipfftExecC2R(testPlan, device_odata, device_idata);
//checkCudaErrors("Performing IFFT");

//hipMemcpy(host_idata, device_idata, outputSize, hipMemcpyDeviceToHost);

//_________________________________________________________________________________________________________


	// ---------------------- Cuda Memory Cleanup ----------------------
	hipfftDestroy(plan);

	return outputElements;
}

unsigned inverseComplexToComplexFFT(hipfftComplex* device_idata, unsigned int inputElements, hipfftComplex **device_odata, int batch)
{
	// ---------------------- Initialise CUDA stuff ---------------------------------
	hipSetDevice(0);
	hipEvent_t event_start, event_stop;
	float timestamp;

	//  Events
	hipEventCreate(&event_start);
	hipEventCreate(&event_stop);

	// FFT size.
	unsigned int outputElements = inputElements;
	unsigned int outputSize = outputElements * sizeof(hipfftComplex) * batch;

	// Init variables.
	hipfftHandle plan;
	hipfftComplex *odata;
	hipMalloc((void **) &odata, outputSize);
	checkCudaErrors("Allocating memory on device");

	// ---------------------- FFT all the channels in place ----------------------
	hipfftPlan1d(&plan, inputElements, HIPFFT_C2C, batch);

	hipEventRecord(event_start, 0);
	hipfftExecC2C(plan, device_idata, odata,HIPFFT_BACKWARD);
	hipDeviceSynchronize();
	hipEventRecord(event_stop, 0);
	hipEventSynchronize(event_stop);
	hipEventElapsedTime(&timestamp, event_start, event_stop);
	checkCudaErrors("Performing FFT");
	//printf("Performed forward FFT: %lf\n", timestamp);


	*device_odata = odata;

//_________________________________________________________________________________________________________
// TESTING: Perform IFFT.
//hipfftHandle testPlan;
//hipfftPlan1d(&testPlan, inputSamples, HIPFFT_C2R, 1);
//hipfftExecC2R(testPlan, device_odata, device_idata);
//checkCudaErrors("Performing IFFT");

//hipMemcpy(host_idata, device_idata, outputSize, hipMemcpyDeviceToHost);

//_________________________________________________________________________________________________________


	// ---------------------- Cuda Memory Cleanup ----------------------
	hipfftDestroy(plan);

	return outputElements;
}
